#include "hip/hip_runtime.h"
// This file is part of ScaViSLAM.
//
// Copyright 2011 Hauke Strasdat (Imperial College London)
//
// ScaViSLAM is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published
// by the Free Software Foundation, either version 3 of the License, or
// any later version.
//
// ScaViSLAM is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with ScaViSLAM.  If not, see <http://www.gnu.org/licenses/>.

#include "dense_tracking.cuh"
#include <stdio.h>

// Many thanks to Steven Lovegrove for various dicussions/comments and hints
// about efficient programming in CUDA!

inline __device__ __host__ float
dotStride3(const float * m_colmajor, const float4 & v1)
{
  return v1.x*m_colmajor[0] + v1.y*m_colmajor[3]
      + v1.z*m_colmajor[6] + v1.w*m_colmajor[9];
}

inline __device__ __host__ float
dotStride4(const float * m_colmajor, const float4 & v1)
{
  return v1.x*m_colmajor[0] + v1.y*m_colmajor[4]
      + v1.z*m_colmajor[8] + v1.w*m_colmajor[12];
}

inline __device__ __host__ float4
matTimesVec(const GpuMatrix4 & m, const float4 & v)
{
  return make_float4(dotStride4(m.data_colmajor,   v),
                     dotStride4(m.data_colmajor+1, v),
                     dotStride4(m.data_colmajor+2, v),
                     dotStride4(m.data_colmajor+3, v));
}

inline __device__ __host__ float4
matTimesVec(const GpuMatrix34 & m, const float4 & v)
{
  return make_float4(dotStride3(m.data_colmajor,   v),
                     dotStride3(m.data_colmajor+1, v),
                     dotStride3(m.data_colmajor+2, v),
                     1.f);
}

inline __device__ __host__ float2
cameraProject(const GpuIntrinsics & intrinsics, const float4 & p)
{
  return make_float2(intrinsics.focal_length*p.x/p.z
                     + intrinsics.principal_point.x,
                     intrinsics.focal_length*p.y/p.z
                     + intrinsics.principal_point.y);
}

inline __device__ __host__ void
frameJacobian(const float4 & p, float focal_length, float dx, float dy,
              GpuVector6 * jac)
{
  float z_sq = p.z*p.z;

  dx *= focal_length;
  dy *= focal_length;

  jac->data[0] = -dx*(1./p.z);
  jac->data[1] = -dy*1./p.z;
  jac->data[2] = (dx*p.x/z_sq + dy*p.y/z_sq);
  jac->data[3] = (dx*(p.x*p.y)/z_sq + dy*(1.f+p.y*p.y/z_sq));
  jac->data[4] = (-dx*(1.f+(p.x*p.x/z_sq))-dy*(p.x*p.y)/z_sq);
  jac->data[5] = (dx*p.y/p.z-dy*p.x/p.z);
}

__global__ void
pointcloud_kernel(GpuMatrix4 TQ,
                  const float * disparities,
                  int width,
                  int height,
                  int stride_in,
                  int stride_out,
                  int factor,
                  float4 * point_cloud)
{
  int u = (blockIdx.x*blockDim.x + threadIdx.x);
  int v = (blockIdx.y*blockDim.y + threadIdx.y);

  if (u<width && v<height)
  {
    int x = u*factor;
    int idx_in = v*stride_in + x;
    int idx_out = v*stride_out + u;

    float4 point;
    float d = disparities[idx_in]*factor;
    if (d<=0)
    {
      point = make_float4(0.f, 0.f, 0.f, -1.f);
    }
    else
    {
      //      if (d==0)
      //      {
      //        d = 0.00000000001f;
      //      }
      float4 uvd = make_float4(u, v, d, 1.f);
      point = matTimesVec(TQ,uvd);
      point.x /= point.w;
      point.y /= point.w;
      point.z /= point.w;
      point.w = 1.f;
    }
    point_cloud[idx_out] = point;
  }
}

void
computePointCloud(const GpuMatrix4 & TQ_actkey_from_cur,
                  const float * disparties,
                  int width,
                  int height,
                  int stride_in,
                  int stride_out,
                  int factor,
                  float4 * point_cloud)
{
  dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  dim3 grid_size((width + block_size.x - 1)/ block_size.x,
                 (height + block_size.y - 1) / block_size.y, 1);

  pointcloud_kernel
      <<<grid_size, block_size>>>(TQ_actkey_from_cur,
                                  disparties,
                                  width,
                                  height,
                                  stride_in,
                                  stride_out,
                                  factor,
                                  point_cloud);

}

texture<float, 2, hipReadModeElementType> tex_image_cur;
texture<float, 2, hipReadModeElementType> tex_dx_cur;
texture<float, 2, hipReadModeElementType> tex_dy_cur;

template<int block_size>
__device__ __host__ void
warpReduce(int thread_id, volatile GpuTrackingData * shared_tracking_data)
{
  if (block_size>=64)
    shared_tracking_data[thread_id].add(shared_tracking_data[thread_id+32]);
  if (block_size>=32)
    shared_tracking_data[thread_id].add(shared_tracking_data[thread_id+16]);
  if (block_size>=16)
    shared_tracking_data[thread_id].add(shared_tracking_data[thread_id+8]);
  if (block_size>=8)
    shared_tracking_data[thread_id].add(shared_tracking_data[thread_id+4]);
  if (block_size>=4)
    shared_tracking_data[thread_id].add(shared_tracking_data[thread_id+2]);
  if (block_size>=2)
    shared_tracking_data[thread_id].add(shared_tracking_data[thread_id+1]);
}

template<int block_size>
__global__ void
jacobianReduction_kernel(const float * img_prev,
                          const float4 * point_cloud_prev,
                          GpuMatrix34  T_cur_from_prev,
                          GpuIntrinsics intrinsics,
                          int width,
                          int height,
                          int stride_float_img,
                          int stride_float4_img,
                          GpuTrackingData * global_tracking_data)
{
  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;
  int block_id = blockIdx.y*gridDim.x + blockIdx.x;
  int u_prev = blockIdx.x*blockDim.x + threadIdx.x;
  int v_prev = blockIdx.y*blockDim.y + threadIdx.y;
  int idx_float_img = v_prev*stride_float_img + u_prev;
  int idx_float4_img = v_prev*stride_float4_img + u_prev;

  __shared__ GpuTrackingData shared_tracking_data[BLOCK_SiZE];

  shared_tracking_data[thread_id].setZero();
  float4 xyz_prev = point_cloud_prev[idx_float4_img];

  if (u_prev<width && v_prev<height)
  {
    if( xyz_prev.w > 0 )
    {
      float4 xyz_cur = matTimesVec(T_cur_from_prev,xyz_prev);
      float2 uv_cur = cameraProject(intrinsics, xyz_cur);

      if(uv_cur.x >= 1.f && uv_cur.y >= 1.f
         && uv_cur.x <= (float)(width-2) && uv_cur.y <= (float)(height-2))
      {
        float2 uv_cur_texoffset = make_float2(uv_cur.x+0.5f, uv_cur.y+0.5f);

        float intensity_prev = img_prev[idx_float_img];
        float intensity_cur
            = tex2D(tex_image_cur, uv_cur_texoffset.x, uv_cur_texoffset.y);
        //TODO: variable kernel size of derivatives and different factor
        float dx
            = 0.5f*tex2D(tex_dx_cur, uv_cur_texoffset.x, uv_cur_texoffset.y);
        float dy
            = 0.5f*tex2D(tex_dy_cur, uv_cur_texoffset.x, uv_cur_texoffset.y);

        float res  = (intensity_prev-intensity_cur);
        GpuVector6 jacobian;
        frameJacobian(xyz_cur, intrinsics.focal_length, dx, dy, &jacobian);

        shared_tracking_data[thread_id]
            .jacobian_times_res.scaledAdd(jacobian,res);
        shared_tracking_data[thread_id]
            .hessian.addOuter(jacobian);
      }
    }

    // reduction
    __syncthreads();

    if (block_size>=512)
    {
      if (thread_id < 256)
        shared_tracking_data[thread_id]
            .add(shared_tracking_data[thread_id + 256]);
      __syncthreads();
    }
    if (block_size>=256)
    {
      if (thread_id < 128)
        shared_tracking_data[thread_id]
            .add(shared_tracking_data[thread_id + 128]);
      __syncthreads();
    }
    if (block_size>=128)
    {
      if (thread_id < 64)
        shared_tracking_data[thread_id]
            .add(shared_tracking_data[thread_id + 64]);
      __syncthreads();
    }

    if (thread_id<32)
    {
      warpReduce<block_size>(thread_id, shared_tracking_data);
    }

    if(thread_id == 0)
    {
      global_tracking_data[block_id] = shared_tracking_data[0];
    }
  }
}

GpuTracker::GpuTracker(int width, int height)
{
  mem_size_ = ((width+BLOCK_WIDTH-1)/BLOCK_WIDTH)
      * ((height+BLOCK_WIDTH-1)/BLOCK_WIDTH);
  gpu_tracking_data_ = NULL;
  tracking_data_ = NULL;
  gpu_chi2_ = NULL;
  chi2_ = NULL;

  hipMalloc(&gpu_tracking_data_, sizeof(GpuTrackingData)*mem_size_);
  tracking_data_ = new GpuTrackingData[mem_size_];

  hipMalloc(&gpu_chi2_, sizeof(float)*mem_size_);
  chi2_ = new float[mem_size_];

  assert(gpu_tracking_data_!=NULL);
  assert(tracking_data_!=NULL);
  assert(gpu_chi2_!=NULL);
  assert(chi2_!=NULL);

  tex_image_cur.filterMode = hipFilterModeLinear;
  tex_dx_cur.filterMode = hipFilterModeLinear;
  tex_dy_cur.filterMode = hipFilterModeLinear;

  float_descriptor_ =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
}

GpuTracker::~GpuTracker()
{
  hipFree(gpu_tracking_data_);
  delete[] tracking_data_;
  hipFree(gpu_chi2_);
  delete[] chi2_;
}

void GpuTracker::
bindTexture(const float * img_cur,
            const float * dx_img_cur,
            const float * dy_img_cur,
            int width,
            int height,
            int stride_float_img)
{
  hipBindTexture2D(0, tex_image_cur, img_cur, float_descriptor_,
                    width, height, stride_float_img*sizeof(float));
  hipBindTexture2D(0, tex_dx_cur, dx_img_cur, float_descriptor_,
                    width, height, stride_float_img*sizeof(float));
  hipBindTexture2D(0, tex_dy_cur, dy_img_cur,float_descriptor_,
                    width, height, stride_float_img*sizeof(float));
}


void GpuTracker::
jacobianReduction(const float * img_prev,
                   const float4* point_cloud_prev,
                   const GpuMatrix34 & T_cur_from_prev,
                   const GpuIntrinsics & intrinsics,
                   int width,
                   int height,
                   int stride_float_img,
                   int stride_float4_img,
                   GpuTrackingData * tracking_result)
{
  const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  dim3 grid_size((width + block_size.x-1)/ block_size.x,
                 (height + block_size.y-1) / block_size.y, 1);

  jacobianReduction_kernel<BLOCK_SiZE>
      <<<grid_size, block_size>>>(img_prev,
                                  point_cloud_prev,
                                  T_cur_from_prev,
                                  intrinsics,
                                  width,
                                  height,
                                  stride_float_img,
                                  stride_float4_img,
                                  gpu_tracking_data_);
  hipDeviceSynchronize();

  hipMemcpy(tracking_data_, gpu_tracking_data_,
             sizeof(GpuTrackingData)*grid_size.x*grid_size.y,
             hipMemcpyDeviceToHost);

  tracking_result->jacobian_times_res.setZero();
  tracking_result->hessian.setZero();

  int global_mem_size = grid_size.x*grid_size.y;
  assert(mem_size_ >= global_mem_size);
  for (int i=0; i<global_mem_size; ++i)
    tracking_result->add(tracking_data_[i]);
}

template<int block_size>
__device__ __host__ void
warpReduce(int thread_id, volatile float * shared_float)
{
  if (block_size>=64)
    shared_float[thread_id] += shared_float[thread_id+32];
  if (block_size>=32)
    shared_float[thread_id] += shared_float[thread_id+16];
  if (block_size>=16)
    shared_float[thread_id] += shared_float[thread_id+8];
  if (block_size>=8)
    shared_float[thread_id] += shared_float[thread_id+4];
  if (block_size>=4)
    shared_float[thread_id] += shared_float[thread_id+2];
  if (block_size>=2)
    shared_float[thread_id] += shared_float[thread_id+1];
}

template<int block_size>
__global__ void
chi2_kernel(const float * img_prev,
            const float4 * point_cloud_prev,
            GpuMatrix34  T_cur_from_prev,
            GpuIntrinsics intrinsics,
            int width,
            int height,
            int stride_float_img,
            int stride_float4_img,
            float * global_chi2)
{
  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;
  int block_id = blockIdx.y*gridDim.x + blockIdx.x;
  int u_prev = blockIdx.x*blockDim.x + threadIdx.x;
  int v_prev = blockIdx.y*blockDim.y + threadIdx.y;
  int idx_float_img = v_prev*stride_float_img + u_prev;
  int idx_float4_img = v_prev*stride_float4_img + u_prev;

  __shared__ float shared_chi2[BLOCK_WIDTH*BLOCK_WIDTH];

  shared_chi2[thread_id] = 0.f;
  float4 xyz_prev = point_cloud_prev[idx_float4_img];

  if (u_prev<width && v_prev<height)
  {
    if( xyz_prev.w > 0 )
    {
      float4 xyz_cur = matTimesVec(T_cur_from_prev,xyz_prev);
      float2 uv_cur = cameraProject(intrinsics, xyz_cur);

      if(uv_cur.x >= 1.f && uv_cur.y >= 1.f
         && uv_cur.x <= (float)(width-2) && uv_cur.y <= (float)(height-2))
      {
        float2 uv_cur_texoffset = make_float2(uv_cur.x+0.5f, uv_cur.y+0.5f);

        float intensity_prev = img_prev[idx_float_img];
        float intensity_cur
            = tex2D(tex_image_cur, uv_cur_texoffset.x, uv_cur_texoffset.y);
        float res  = (intensity_prev-intensity_cur);
        shared_chi2[thread_id] += res*res;
      }
    }

    // reduction
    __syncthreads();

    if (block_size>=512)
    {
      if (thread_id < 256)
        shared_chi2[thread_id] += shared_chi2[thread_id + 256];
      __syncthreads();
    }
    if (block_size>=256)
    {
      if (thread_id < 128)
        shared_chi2[thread_id] += shared_chi2[thread_id + 128];
      __syncthreads();
    }
    if (block_size>=128)
    {
      if (thread_id < 64)
        shared_chi2[thread_id] += shared_chi2[thread_id + 64];
      __syncthreads();
    }

    // unroll last loop
    if (thread_id<32)
    {
      warpReduce<block_size>(thread_id, shared_chi2);
    }

    if(thread_id == 0)
    {
      global_chi2[block_id] = shared_chi2[0];
    }
  }
}

float GpuTracker::
chi2(const float * img_prev,
     const float4* point_cloud_prev,
     const GpuMatrix34 & T_cur_from_prev,
     const GpuIntrinsics & intrinsics,
     int width,
     int height,
     int stride_float_img,
     int stride_float4_img)
{
  float output_chi2 = 0.f;
  const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  dim3 grid_size((width + block_size.x-1)/ block_size.x,
                 (height + block_size.y-1) / block_size.y, 1);

  chi2_kernel<BLOCK_SiZE>
      <<<grid_size, block_size>>>(img_prev,
                                  point_cloud_prev,
                                  T_cur_from_prev,
                                  intrinsics,
                                  width,
                                  height,
                                  stride_float_img,
                                  stride_float4_img,
                                  gpu_chi2_);
  hipDeviceSynchronize();

  hipMemcpy(chi2_, gpu_chi2_,
             sizeof(float)*grid_size.x*grid_size.y,
             hipMemcpyDeviceToHost);

  int global_mem_size = grid_size.x*grid_size.y;
  assert(mem_size_ >= global_mem_size);
  for (int i=0; i<global_mem_size; ++i)
    output_chi2 += chi2_[i];
  return output_chi2;
}


__global__ void
residualImage_kernel(const float * img_prev,
                     const float4 * point_cloud_prev,
                     GpuMatrix34  T_cur_from_prev,
                     GpuIntrinsics intrinsics,
                     int width,
                     int height,
                     int stride_float_img,
                     int stride_float4_img,
                     float4 * res_img)
{
  int u_prev = blockIdx.x*blockDim.x + threadIdx.x;
  int v_prev = blockIdx.y*blockDim.y + threadIdx.y;
  int idx_float_img = v_prev*stride_float_img + u_prev;
  int idx_float4_img = v_prev*stride_float4_img + u_prev;

  float4 xyz_prev = point_cloud_prev[idx_float4_img];

  if (u_prev<width && v_prev<height)
  {
    if(xyz_prev.w > 0)
    {
      float4 xyz_cur = matTimesVec(T_cur_from_prev,xyz_prev);
      float2 uv_cur = cameraProject(intrinsics, xyz_cur);

      if(uv_cur.x >= 1.f && uv_cur.y >= 1.f
         && uv_cur.x <= (float)(width-2) && uv_cur.y <= (float)(height-2))
      {
        float2 uv_cur_texoffset = make_float2(uv_cur.x+0.5f, uv_cur.y+0.5f);

        float intensity_prev = img_prev[idx_float_img];
        float intensity_cur
            = tex2D(tex_image_cur, uv_cur_texoffset.x, uv_cur_texoffset.y);
        float res  = (intensity_prev-intensity_cur);
        float v = max(0.f, 1-50.f*res*res);
        res_img[idx_float4_img] = make_float4(v, v, v, 1.f);
      }
      else
      {
        res_img[idx_float4_img] = make_float4(1.f, 0.f, 0.f, 1.f);
      }
    }
    else
    {
      res_img[idx_float4_img] = make_float4(0.f, 1.f, 0.f, 1.f);
    }
  }
}

void GpuTracker::
residualImage(const float * img_prev,
              const float4* point_cloud_prev,
              const GpuMatrix34 & T_cur_from_prev,
              const GpuIntrinsics & intrinsics,
              int width,
              int height,
              int stride_float_img,
              int stride_float4_img,
              float4 * res_img)
{
  const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  dim3 grid_size((width + block_size.x-1)/ block_size.x,
                 (height + block_size.y-1) / block_size.y, 1);

  residualImage_kernel
      <<<grid_size, block_size>>>(img_prev,
                                  point_cloud_prev,
                                  T_cur_from_prev,
                                  intrinsics,
                                  width,
                                  height,
                                  stride_float_img,
                                  stride_float4_img,
                                  res_img);
  hipDeviceSynchronize();
}
